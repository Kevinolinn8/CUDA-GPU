#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <cmath>

#include "util.h"

const int BLOCKSIZE = 32;
constexpr float THRESHOLD = 1e-6f;



__global__ void spmv(const int *row_ptr, const int *col_ind, const float *vals, const float *x, float *y) {

    int myThread = threadIdx.x;
    int block = blockDim.x;
    int row = blockIdx.x;

    __shared__ float sum[BLOCKSIZE];

    int start = row_ptr[row];
    int end = row_ptr[row + 1];
    

    float temp = 0.f;
    for(int i = start + myThread; i < end; i+=block){
        temp += vals[i] * x[col_ind[i]];
    }

    sum[myThread] = temp;
    for(int stride = block/2; stride > 0; stride >>= 1){

        __syncthreads();
        
        if(myThread < stride){
           sum[myThread] += sum[myThread + stride];
        }
    }
    __syncthreads();

    if(myThread == 0){ 
    y[row] = sum[0];
    }
}



float * serial_implementation(float * sparse_matrix, int * ptr, int * indices, float * dense_vector, int rows) {
    float * output = (float *)malloc(sizeof(float) * rows);
    
    for (int i = 0; i < rows; i++) {
        float accumulator = 0.f;
        for (int j = ptr[i]; j < ptr[i+1]; j++) {
            accumulator += sparse_matrix[j] * dense_vector[indices[j]];
        }
        output[i] = accumulator;
    }
    
    return output;
}

int main(int argc, char ** argv) {
    
    assert(argc == 2);
    
    // input_cpu
    float * sparse_matrix = nullptr; 
    float * dense_vector = nullptr;
    int * ptr = nullptr;
    int * indices = nullptr;
    int values = 0, rows = 0, cols = 0;

    

    //input_gpu
    float * matrix_gpu = nullptr;
    float * vector_gpu = nullptr;
    int * ptr_gpu = nullptr;
    int * indices_gpu = nullptr;
    
    
    read_sparse_file(argv[1], &sparse_matrix, &ptr, &indices, &values, &rows, &cols);
    printf("%d %d %d\n", values, rows, cols);
    dense_vector = (float *)malloc(sizeof(float) * cols);

    // Generate "random" vector
    std::mt19937 gen(13); // Keep constant to maintain determinism between runs
    std::uniform_real_distribution<> dist(-10.0f, 10.0f);
    for (int i = 0; i < cols; i++) {
        dense_vector[i] = dist(gen);
    }

    hipStream_t stream;
    hipEvent_t begin, end;
    hipStreamCreate(&stream);
    hipEventCreate(&begin);
    hipEventCreate(&end);

    float * h_output = (float *)malloc(sizeof(float) * rows); // THIS VARIABLE SHOULD HOLD THE TOTAL COUNT BY THE END

    

    // PERFORM NECESSARY VARIABLE DECLARATIONS HERE
    //int * data = read_file(argv[1], &rows, &cols);
    float * retarray = nullptr;
    
     //allocate memory
    hipMalloc(&matrix_gpu, sizeof(float) * values);
    hipMalloc(&vector_gpu, sizeof(float) * cols);
    hipMalloc(&ptr_gpu, sizeof(int) * (rows + 1));
    hipMalloc(&indices_gpu, sizeof(int) * values);
    hipMalloc(&retarray, sizeof(float) * rows);



    //PERFORM NECESSARY DATA TRANSFER HERE
    hipMemcpyAsync(matrix_gpu, sparse_matrix, sizeof(float) * values, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(ptr_gpu, ptr, sizeof(int) * (rows + 1), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(vector_gpu, dense_vector, sizeof(float) * cols, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(indices_gpu, indices, sizeof(int) * values, hipMemcpyHostToDevice, stream);


    // LAUNCH KERNEL HERE
    dim3 block(BLOCKSIZE);
    dim3 grid(rows);
    hipEventRecord(begin, stream);
    spmv<<<grid,block,0,stream>>>(ptr_gpu,indices_gpu, matrix_gpu, vector_gpu, retarray);
    hipEventRecord(end, stream);
 

   // PERFORM NECESSARY DATA TRANSFER HERE
    hipMemcpyAsync(h_output, retarray, sizeof(float) * rows, hipMemcpyDeviceToHost, stream);


    hipStreamSynchronize(stream);

    float ms;
    hipEventElapsedTime(&ms, begin, end);
    printf("Elapsed time: %f ms\n", ms);

    float * reference_output = serial_implementation(sparse_matrix, ptr, indices, dense_vector, rows);
    for (int i = 0; i < rows; i++) {
        if (fabs(reference_output[i] - h_output[i]) > THRESHOLD) {
            printf("ERROR: %f != %f at index %d\n", reference_output[i], h_output[i], i);
            abort();
        }
    }

    hipEventDestroy(begin);
    hipEventDestroy(end);
    hipStreamDestroy(stream);


    //FREE THE VARIABLES I USED
    hipFree(matrix_gpu);
    hipFree(vector_gpu);
    hipFree(ptr_gpu);
    hipFree(indices_gpu);
    hipFree(retarray);
    free(reference_output);
    free(h_output);
    hipEventDestroy(begin);
    hipEventDestroy(end);
    hipStreamDestroy(stream);

    free(sparse_matrix);
    free(dense_vector);
    free(ptr);
    free(indices);
    free(reference_output);
    free(h_output);

    return 0;
}   

